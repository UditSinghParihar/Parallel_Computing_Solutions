#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int* a, int* b, int* c){
	*c = *a + *b;
	//printf("c = %d\n", *c);
	printf("Thread id: %d\n", threadIdx.x);
	printf("Block id:%d\n", blockIdx.x);
}

int main(void) {
	int a, b, c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = sizeof(int);
	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Setup input values
	a = 2;
	b = 7;

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU
	add<<<3,5>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();
	// Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	//printf("a+b = %d\n", c);
	// Cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
