#include <stdio.h>
#include <hip/hip_runtime.h>

#include <stdlib.h>

#define N 256

__global__ void add(int *a, int *b, int *c) {
	int tx = threadIdx.x, ty = threadIdx.y;
	int bidx = blockIdx.x, bidy = blockIdx.y;
	int bx = blockDim.x, by = blockDim.y;
	int gx = gridDim.x, gy = gridDim.y;
	
	int bid = bidx*gy + bidy;	
	int lid = tx*by + ty;
	int tid = bid*bx*by + lid;
	int dim = 16;
	c[tid] = a[tid] + b[tid];
	
__shared__ int blocksum[16];
	blocksum[lid] = a[tid] + b[tid];
	__syncthreads();
	if(tx==0&&ty==0){
		int sum=0;
		for(int i=0;i<dim;++i)sum+=blocksum[i];
			printf("block %d %d sum %d\n",bidx,bidy,sum);
	}
}

void print_five(int* a, int* b, int* c){
	int r = 25;
	for(int i=0; i<10; ++i){
		r = rand()%N;	
		printf("%d %d %d\n", a[r], b[r], c[r]);
	}
}

void random_ints(int *a, int n){
   int i;
   for (i = 0; i < n; ++i)
       a[i] = rand() %10;
}

int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU with N blocks
	
	dim3 thread(4,4),block(4, 4);

	add<<<block, thread>>>(d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	print_five(a,b,c);

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
