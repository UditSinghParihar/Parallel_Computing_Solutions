#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "dvec.h"

using namespace std;

#define BlockMax 65535
#define ThreadMax 1024

__global__ void add(float *x, float *y, int n){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	int stride = blockDim.x*gridDim.x;

	// if(index==0)
	// 	printf("Blocks: %d\tThreads: %d\n--\n", gridDim.x, blockDim.x);
	
	for(int i=index; i<n; i+=stride)
		y[i] += x[i]; 
}

void print_vector(const vector<float>& vec){
	for(auto element : vec){;
		fprintf(stdout, "%9.4f\n", element);
	}
	cout << "\n---\n";
}

int main(int argc, char const *argv[]){
	const int N = 10;
	vector<float> x(N);
	vector<float> y(N);

	for(int i=0; i<x.size(); ++i){
		x[i] = i;
		y[i] = 2.0*i;
	}
	print_vector(x);
	print_vector(y);

	// float *d_x, *d_y;
	// size_t sz = N*sizeof(float);
	// hipMalloc(&d_x, sz);
	// hipMalloc(&d_y, sz);
	// hipMemcpy(d_x, x.data(), sz, hipMemcpyHostToDevice);
	// hipMemcpy(d_y, y.data(), sz, hipMemcpyHostToDevice);

	// fprintf(stdout, "Kernel started.\n");
	// add<<<(N+ThreadMax-1)/ThreadMax, ThreadMax>>>(d_x, d_y, N);
	// hipDeviceSynchronize();
	// fprintf(stdout, "Kernel ended. \n");

	// hipMemcpy(y.data(), d_y, sz, hipMemcpyDeviceToHost);
	// // print_vector(y);

	// hipFree(d_x);
	// hipFree(d_y);

	dvec<float> d_x(N);
	dvec<float> d_y(N);

	d_x.set(&x[0]);
	d_y.set(&y[0]);

	add<<<(N+ThreadMax-1)/ThreadMax, ThreadMax>>>(d_x.data(), d_y.data(), N);
	hipDeviceSynchronize();
	d_y.get(&y[0]);

	print_vector(y);	


	return 0;
}